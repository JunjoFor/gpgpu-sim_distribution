#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MAX_THREADS_PER_BLOCK 512

int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
    int starting;
    int no_of_edges;
}; 
__global__ void
Kernel( Node* g_graph_nodes, int* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool *     g_graph_visited, int* g_cost, int no_of_nodes)
{
    int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
    if( tid<no_of_nodes && g_graph_mask[tid])
    {
        g_graph_mask[tid]=false;
        for(int i=g_graph_nodes[tid].starting; i<(g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].       starting); i++)
            {
            int id = g_graph_edges[i];
            if(!g_graph_visited[id])
                {
                g_cost[id]=g_cost[tid]+1;
                g_updating_graph_mask[id]=true;
                }
            }
    }
}

__global__ void
Kernel2( bool* g_graph_mask, bool *g_updating_graph_mask, bool* g_graph_visited, bool *g_over, int           no_of_nodes)
{
    int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
    if( tid<no_of_nodes && g_updating_graph_mask[tid])
    {

        g_graph_mask[tid]=true;
        g_graph_visited[tid]=true;
        *g_over=true;
        g_updating_graph_mask[tid]=false;
    }
}

void BFSGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    no_of_nodes=0;
    edge_list_size=0;
    BFSGraph( argc, argv);
}



////////////////////////////////////////////////////////////////////////////////
//Apply BFS on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
void BFSGraph( int argc, char** argv)
{
       static char *input_file_name;
       static char *goldfile;
       //printf("argc=%d\n", argc);
       if (argc >= 2 ) {
               input_file_name = argv[1];
               goldfile = argv[2];
               printf("Input file: %s\n", input_file_name);
       }
       else
       {
               input_file_name = "SampleGraph.txt";
               printf("No input file specified, defaulting to SampleGraph.txt\n");
       }

    printf("Reading File\n");
        //Read in Graph from a file
    fp = fopen(input_file_name,"r");
    if(!fp)
    {
        printf("Error Reading graph file\n");
        return;
    }

    int source = 0;

    fscanf(fp,"%d",&no_of_nodes);

    int num_of_blocks = 1;
    int num_of_threads_per_block = no_of_nodes;

    //Make execution Parameters according to the number of nodes
    //Distribute threads across multiple Blocks if necessary
    if(no_of_nodes>MAX_THREADS_PER_BLOCK)
    {
        num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK);
        num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
    }

    // allocate host memory
    Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
    bool *h_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
    bool *h_updating_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
    bool *h_graph_visited = (bool*) malloc(sizeof(bool)*no_of_nodes);

    int start, edgeno;
    // initalize the memory
    for( unsigned int i = 0; i < no_of_nodes; i++)
    {
        fscanf(fp,"%d %d",&start,&edgeno);
        h_graph_nodes[i].starting = start;
        h_graph_nodes[i].no_of_edges = edgeno;
        h_graph_mask[i]=false;
        h_updating_graph_mask[i]=false;
        h_graph_visited[i]=false;
    }

    //read the source node from the file
    fscanf(fp,"%d",&source);
    source=0;

    //set the source node as true in the mask
    h_graph_mask[source]=true;
    h_graph_visited[source]=true;

    fscanf(fp,"%d",&edge_list_size);

    int id,cost;
    int* h_graph_edges = (int*) malloc(sizeof(int)*edge_list_size);
    for(int i=0; i < edge_list_size ; i++)
    {
        fscanf(fp,"%d",&id);
        fscanf(fp,"%d",&cost);
        h_graph_edges[i] = id;
    }

    if(fp)
        fclose(fp);

    printf("Read File\n");

    //Copy the Node list to device memory
    Node* d_graph_nodes;
    hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
    hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

    //Copy the Edge List to device Memory
    int* d_graph_edges;
    hipMalloc( (void**) &d_graph_edges, sizeof(int)*edge_list_size) ;
    hipMemcpy( d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;

    //Copy the Mask to device memory
    bool* d_graph_mask;
    hipMalloc( (void**) &d_graph_mask, sizeof(bool)*no_of_nodes) ;
    hipMemcpy( d_graph_mask, h_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;

    bool* d_updating_graph_mask;
    hipMalloc( (void**) &d_updating_graph_mask, sizeof(bool)*no_of_nodes) ;
    hipMemcpy( d_updating_graph_mask, h_updating_graph_mask, sizeof(bool)*no_of_nodes,                      hipMemcpyHostToDevice) ;

    //Copy the Visited nodes array to device memory
    bool* d_graph_visited;
    hipMalloc( (void**) &d_graph_visited, sizeof(bool)*no_of_nodes) ;
    hipMemcpy( d_graph_visited, h_graph_visited, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;

    // allocate mem for the result on host side
    int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
    for(int i=0;i<no_of_nodes;i++)
        h_cost[i]=-1;
    h_cost[source]=0;

    // allocate device memory for result
    int* d_cost;
    hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes);
    hipMemcpy( d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;

    //make a bool to check if the execution is over
    bool *d_over;
    hipMalloc( (void**) &d_over, sizeof(bool));

    printf("Copied Everything to GPU memory\n");

    // setup execution parameters
    dim3  grid( num_of_blocks, 1, 1);
    dim3  threads( num_of_threads_per_block, 1, 1);

    int k=0;

    bool stop;
    //Call the Kernel untill all the elements of Frontier are not false
    do
    {
        //if no thread changes this value then the loop stops
        stop=false;
        hipMemcpy( d_over, &stop, sizeof(bool), hipMemcpyHostToDevice) ;
        Kernel<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_mask, d_updating_graph_mask,   d_graph_visited, d_cost, no_of_nodes);
        // check if kernel execution generated and error


        Kernel2<<< grid, threads, 0 >>>( d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over,       no_of_nodes);
        // check if kernel execution generated and error


        hipMemcpy( &stop, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;
        k++;
    }
    while(stop);


    printf("Kernel Executed %d times\n",k);

    // copy result from device to host
    hipMemcpy( h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) ;

    //Store the result into a file
    FILE *fpo = fopen("result.txt","w");
    for(int i=0;i<no_of_nodes;i++)
        fprintf(fpo,"%d) cost:%d\n",i,h_cost[i]);
    fclose(fpo);
    printf("Result stored in result.txt\n");

    if(goldfile){
        FILE *gold = fopen(goldfile, "r");
        FILE *result = fopen("result.txt", "r");
        int result_error=0;
        while(!feof(gold)&&!feof(result)){
            if (fgetc(gold)!=fgetc(result)) {
                result_error = 1;
                break;
            }
        }
        if((feof(gold)^feof(result)) | result_error) {
            printf("\nFAILED\n");
        } else {
            printf("\nPASSED\n");
        }

        fclose(gold);
        fclose(result);
    }

    // cleanup memory
    free( h_graph_nodes);
    free( h_graph_edges);
    free( h_graph_mask);
    free( h_updating_graph_mask);
    free( h_graph_visited);
    free( h_cost);
    hipFree(d_graph_nodes);
    hipFree(d_graph_edges);
    hipFree(d_graph_mask);
    hipFree(d_updating_graph_mask);
    hipFree(d_graph_visited);
    hipFree(d_cost);
}
